#include "hip/hip_runtime.h"
// --------------------------------------------------------
// Written by Bharat Singh, 2017.
// --------------------------------------------------------
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* a, double b) { return b; }
#endif
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/deformable_psroi_pooling_layer.hpp"
#include "caffe/util/gpu_util.cuh"

using std::max;
using std::min;

namespace caffe {
    template <typename DType>
    __device__ DType bilinear_interp(
      const DType* data,
      const DType x,
      const DType y,
      const int width,
      const int height) {
      int x1 = floor(x);
      int x2 = ceil(x);
      int y1 = floor(y);
      int y2 = ceil(y);
      DType dist_x = static_cast<DType>(x - x1);
      DType dist_y = static_cast<DType>(y - y1);
      DType value11 = data[y1*width + x1];
      DType value12 = data[y2*width + x1];
      DType value21 = data[y1*width + x2];
      DType value22 = data[y2*width + x2];
      DType value = (1 - dist_x)*(1 - dist_y)*value11 + (1 - dist_x)*dist_y*value12
        + dist_x*(1 - dist_y)*value21 + dist_x*dist_y*value22;
      return value;
    }

    template <typename DType>
    __global__ void DeformablePSROIPoolForwardKernel(
      const int count,
      const DType* bottom_data,
      const DType spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const DType* bottom_rois, const DType* bottom_trans,
      const bool no_trans,
      const DType trans_std,
      const int sample_per_part,
      const int output_dim,
      const int group_size,
      const int part_size,
      const int num_classes,
      const int channels_each_class,
      DType* top_data,
      DType* top_count) {
      CUDA_KERNEL_LOOP(index, count) {
        // The output is in order (n, ctop, ph, pw)
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int ctop = (index / pooled_width / pooled_height) % output_dim;
        int n = index / pooled_width / pooled_height / output_dim;

        // [start, end) interval for spatial sampling
        const DType* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
        DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
        DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
        DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

        // Force too small ROIs to be 1x1
        DType roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
        DType roi_height = max(roi_end_h - roi_start_h, 0.1);

        // Compute w and h at bottom
        DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
        DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

        DType sub_bin_size_h = bin_size_h / static_cast<DType>(sample_per_part);
        DType sub_bin_size_w = bin_size_w / static_cast<DType>(sample_per_part);

        int part_h = floor(static_cast<DType>(ph) / pooled_height*part_size);
        int part_w = floor(static_cast<DType>(pw) / pooled_width*part_size);
        int class_id = ctop / channels_each_class;
        DType trans_x = no_trans ? static_cast<DType>(0) :
          bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h)*part_size + part_w] * trans_std;
        DType trans_y = no_trans ? static_cast<DType>(0) :
          bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h)*part_size + part_w] * trans_std;
        
        DType wstart = static_cast<DType>(pw)* bin_size_w
          + roi_start_w;
        wstart += trans_x * roi_width;
        DType hstart = static_cast<DType>(ph) * bin_size_h
          + roi_start_h;
        hstart += trans_y * roi_height;
        
        DType sum = 0;
        int count = 0;
        int gw = floor(static_cast<DType>(pw) * group_size / pooled_width);
        int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
        gw = min(max(gw, 0), group_size - 1);
        gh = min(max(gh, 0), group_size - 1);

        const DType* offset_bottom_data = bottom_data + (roi_batch_ind * channels) * height * width;
        for (int ih = 0; ih < sample_per_part; ih++) {
          for (int iw = 0; iw < sample_per_part; iw++) {
            DType w = wstart + iw*sub_bin_size_w;
            DType h = hstart + ih*sub_bin_size_h;
            // bilinear interpolation
            if (w<-0.5 || w>width - 0.5 || h<-0.5 || h>height - 0.5) {
              continue;
            }
            w = min(max(w, 0.), width - 1.);
            h = min(max(h, 0.), height - 1.);
            int c = (ctop*group_size + gh)*group_size + gw;
            DType val = bilinear_interp(offset_bottom_data + c*height*width, w, h, width, height);
            sum += val;
            count++;
          }
        }
        top_data[index] = count == 0 ? static_cast<DType>(0) : sum / count;
        top_count[index] = count;
      }
    }

  template <typename Dtype>
  void DeformablePSROIPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    const Dtype *bottom_trans = no_trans_ ? NULL : bottom[2]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    Dtype* mapping_channel_ptr = mapping_channel_.mutable_gpu_data();
    int count = top[0]->count();
    const int num_classes = no_trans_ ? 1 : bottom[2]->channels()/ 2;
    const int channels_each_class = no_trans_ ? output_dim_ : output_dim_ / num_classes;
    caffe_gpu_set(count, Dtype(0), top_data);
    caffe_gpu_set(count, Dtype(0), mapping_channel_ptr);
    // NOLINT_NEXT_LINE(whitespace/operators)
/*
    PSROIPoolingForward<Dtype> << <CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS >> >(count, bottom_data, spatial_scale_,
      channels_, height_, width_, pooled_height_,
      pooled_width_, bottom_rois, output_dim_, group_size_,
      top_data, mapping_channel_ptr);
*/
     DeformablePSROIPoolForwardKernel<Dtype> << <CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>> >(
        count, bottom_data, spatial_scale_, channels_, height_, width_, pooled_height_, pooled_width_,
        bottom_rois, bottom_trans, no_trans_, trans_std_, sample_per_part_, output_dim_, 
        group_size_, part_size_, num_classes, channels_each_class, top_data, mapping_channel_ptr);
    CUDA_POST_KERNEL_CHECK;
  }
    template <typename DType>
    __global__ void DeformablePSROIPoolBackwardAccKernel(
      const int count,
      const DType* top_diff,
      const DType* top_count,
      const int num_rois,
      const DType spatial_scale,
      const int channels,
      const int height, const int width,
      const int pooled_height, const int pooled_width,
      const int output_dim,
      DType* bottom_data_diff, DType* bottom_trans_diff,
      const DType* bottom_data,
      const DType* bottom_rois,
      const DType* bottom_trans,
      const bool no_trans,
      const DType trans_std,
      const int sample_per_part,
      const int group_size,
      const int part_size,
      const int num_classes,
      const int channels_each_class) {
      CUDA_KERNEL_LOOP(index, count) {
        // The output is in order (n, ctop, ph, pw)
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int ctop = (index / pooled_width / pooled_height) % output_dim;
        int n = index / pooled_width / pooled_height / output_dim;

        // [start, end) interval for spatial sampling
        const DType* offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        DType roi_start_w = static_cast<DType>(round(offset_bottom_rois[1])) * spatial_scale - 0.5;
        DType roi_start_h = static_cast<DType>(round(offset_bottom_rois[2])) * spatial_scale - 0.5;
        DType roi_end_w = static_cast<DType>(round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
        DType roi_end_h = static_cast<DType>(round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

        // Force too small ROIs to be 1x1
        DType roi_width = max(roi_end_w - roi_start_w, 0.1); //avoid 0
        DType roi_height = max(roi_end_h - roi_start_h, 0.1);

        // Compute w and h at bottom
        DType bin_size_h = roi_height / static_cast<DType>(pooled_height);
        DType bin_size_w = roi_width / static_cast<DType>(pooled_width);

        DType sub_bin_size_h = bin_size_h / static_cast<DType>(sample_per_part);
        DType sub_bin_size_w = bin_size_w / static_cast<DType>(sample_per_part);

        int part_h = floor(static_cast<DType>(ph) / pooled_height*part_size);
        int part_w = floor(static_cast<DType>(pw) / pooled_width*part_size);
        int class_id = ctop / channels_each_class;
        DType trans_x = no_trans ? static_cast<DType>(0) :
          bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h)*part_size + part_w] * trans_std;
        DType trans_y = no_trans ? static_cast<DType>(0) :
          bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h)*part_size + part_w] * trans_std;

        DType wstart = static_cast<DType>(pw)* bin_size_w
          + roi_start_w;
        wstart += trans_x * roi_width;
        DType hstart = static_cast<DType>(ph) * bin_size_h
          + roi_start_h;
        hstart += trans_y * roi_height;

        if (top_count[index] <= 0) {
          continue;
        }
        DType diff_val = top_diff[index] / top_count[index];
        const DType* offset_bottom_data = bottom_data + roi_batch_ind * channels * height * width;
        DType* offset_bottom_data_diff = bottom_data_diff + roi_batch_ind * channels * height * width;
        int gw = floor(static_cast<DType>(pw)* group_size / pooled_width);
        int gh = floor(static_cast<DType>(ph)* group_size / pooled_height);
        gw = min(max(gw, 0), group_size - 1);
        gh = min(max(gh, 0), group_size - 1);

        for (int ih = 0; ih < sample_per_part; ih++) {
          for (int iw = 0; iw < sample_per_part; iw++) {
            DType w = wstart + iw*sub_bin_size_w;
            DType h = hstart + ih*sub_bin_size_h;
            // bilinear interpolation
            if (w<-0.5 || w>width - 0.5 || h<-0.5 || h>height - 0.5) {
              continue;
            }
            w = min(max(w, 0.), width - 1.);
            h = min(max(h, 0.), height - 1.);
            int c = (ctop*group_size + gh)*group_size + gw;
            // backward on feature
            int x0 = floor(w);
            int x1 = ceil(w);
            int y0 = floor(h);
            int y1 = ceil(h);
            DType dist_x = w - x0, dist_y = h - y0;
            DType q00 = (1 - dist_x)*(1 - dist_y);
            DType q01 = (1 - dist_x)*dist_y;
            DType q10 = dist_x*(1 - dist_y);
            DType q11 = dist_x*dist_y;
            int bottom_index_base = c * height *width;
            atomicAdd(offset_bottom_data_diff + bottom_index_base + y0*width + x0, q00*diff_val);
            atomicAdd(offset_bottom_data_diff + bottom_index_base + y1*width + x0, q01*diff_val);
            atomicAdd(offset_bottom_data_diff + bottom_index_base + y0*width + x1, q10*diff_val);
            atomicAdd(offset_bottom_data_diff + bottom_index_base + y1*width + x1, q11*diff_val);

            if (no_trans) {
              continue;
            }
            DType U00 = offset_bottom_data[bottom_index_base + y0*width + x0];
            DType U01 = offset_bottom_data[bottom_index_base + y1*width + x0];
            DType U10 = offset_bottom_data[bottom_index_base + y0*width + x1];
            DType U11 = offset_bottom_data[bottom_index_base + y1*width + x1];
            DType diff_x = (U11*dist_y + U10*(1 - dist_y) - U01*dist_y - U00*(1 - dist_y))
              *trans_std*diff_val;
            diff_x *= roi_width;
            DType diff_y = (U11*dist_x + U01*(1 - dist_x) - U10*dist_x - U00*(1 - dist_x))
              *trans_std*diff_val;
            diff_y *= roi_height;

            atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2) * part_size + part_h)*part_size + part_w, diff_x);
            atomicAdd(bottom_trans_diff + (((n * num_classes + class_id) * 2 + 1)*part_size + part_h)*part_size + part_w, diff_y);
          }
        }
      }
    }

  template <typename Dtype>
  void DeformablePSROIPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
    if (!propagate_down[0]) {
      return;
    }
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_rois = bottom[1]->gpu_data();
    
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype *bottom_trans = no_trans_ ? NULL : bottom[2]->gpu_data();
    Dtype* bottom_data_diff = bottom[0]->mutable_gpu_diff();
    Dtype *bottom_trans_diff = no_trans_ ? NULL : bottom[2]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    const Dtype* mapping_channel_ptr = mapping_channel_.gpu_data();
    const int num_classes = no_trans_ ? 1 : bottom[2]->channels()/ 2;
    const int channels_each_class = no_trans_ ? output_dim_ : output_dim_ / num_classes;
    caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
    if(!no_trans_)
    	caffe_gpu_set(bottom[2]->count(), Dtype(0), bottom[2]->mutable_gpu_diff());
    caffe_gpu_set(bottom_count, Dtype(0), bottom_data_diff);
    const int count = top[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    DeformablePSROIPoolBackwardAccKernel<Dtype> << <CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>> >(
        count, top_diff, mapping_channel_ptr, top[0]->num(), spatial_scale_, channels_, height_, width_,
        pooled_height_, pooled_width_, output_dim_, bottom_data_diff, bottom_trans_diff,
        bottom_data, bottom_rois, bottom_trans, no_trans_, trans_std_, sample_per_part_,
        group_size_, part_size_, num_classes, channels_each_class);
    CUDA_POST_KERNEL_CHECK;
  }

  INSTANTIATE_LAYER_GPU_FUNCS(DeformablePSROIPoolingLayer);

}  // namespace caffe
