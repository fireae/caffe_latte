#include "hip/hip_runtime.h"
// --------------------------------------------------------
// Multitask Network Cascade
// Written by Haozhi Qi
// Copyright (c) 2016, Haozhi Qi
// Licensed under The MIT License [see LICENSE for details]
// --------------------------------------------------------

#include "caffe/layers/mask_pooling_layer.hpp"
#include <iostream>
namespace caffe {

template <typename Dtype>
__global__ void MaskPoolingForward(const int nthreads, const Dtype* bottom_data, const Dtype* bottom_masks,
			       Dtype* top_data, const int channels, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the top output
    int pw = index % width;
    int ph = (index / width) % height;
    // int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int mask_index = n * height * width + ph * width + pw;

    // top feature map has identical shape with bottom feature map, so we reuse index here
    top_data[index] = bottom_data[index] * bottom_masks[mask_index];
  }
}

template <typename Dtype>
void MaskPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
					   const vector<Blob<Dtype>*>& top) {
  // bottom[0] is feature maps, of shape (n x c x h x w)
  // bottom[1] is masks, of shape (n x 1 x h x w)
  // output(n, c, h, w) = input_feature(n, c, h, w) * input_mask(n, 1, h, w)
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_masks = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  MaskPoolingForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >>> 
  (count, bottom_data, bottom_masks, top_data, channels_, height_, width_);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void MaskPoolingBackwardFeature(const int nthreads, const Dtype* bottom_data, const Dtype* bottom_masks,
        Dtype* bottom_diff, const Dtype* top_diff, const int channels, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, h, w) coords in bottom data
    int w = index % width;
    int h = (index / width) % height;
    // int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    
    // output w,h coordinate has the same size with input's w,h coordinate
    int mask_index = n * height * width + h * width + w;
    Dtype float_mask = bottom_masks[mask_index];
    bottom_diff[index] = top_diff[index] * float_mask;
  }
}

template <typename Dtype>
__global__ void MaskPoolingBackwardMask(const int nthreads, const Dtype* bottom_data, Dtype* bottom_diff, 
  const Dtype* top_diff, const int channels, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, w, h) are index of mask element, with channel dim = 1
    int w = index % width;
    int h = (index / width) % height;
    int n = index / width / height / 1;
    Dtype gradient = 0.0;
    for (int i = 0; i < channels; ++i) {
      int data_index = ((n * channels + i) * height + h) * width + w;
      gradient += top_diff[data_index] * bottom_data[data_index];
    }
    int mask_index = ((n * height) + h) * width + w;
    bottom_diff[mask_index] = gradient;
  }
}

template <typename Dtype>
void MaskPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
					    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_masks = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  
  if (propagate_down[0]) {
    MaskPoolingBackwardFeature<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >>>
      (count, bottom_data, bottom_masks, bottom_diff, top_diff, channels_, height_, width_);
  }
  Dtype* bottom_mask_diff = bottom[1]->mutable_gpu_diff();
  count = bottom[1]->count();
  if (propagate_down[1]) {
    MaskPoolingBackwardMask<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >>>
      (count, bottom_data, bottom_mask_diff, top_diff, channels_, height_, width_);
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(MaskPoolingLayer);

} // namespace caffe
